#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
using namespace std;

#include "get_time.h"

typedef float Real;

string realTypename(float x) { return "float"; }
string realTypename(double x) { return "double"; }

__global__
void calculate (const int logN, const int logNB, Real *ma, Real *mb, Real *mc) {
  const int n = 1<<logN;
  const int mask1  = (1<<logNB)-1;
  const int shift2 = logNB;
  const int mask2  = ((1<<logNB)-1)<<logNB;
  const int shift3 = logNB;
  const int mask3  = ((1<<(logN-logNB))-1)<<(2*logNB);
  const int shift4 = logN;
  const int mask4  = ((1<<(logN-logNB))-1)<<(logNB+logN);
  
  for (int addr = blockIdx.x * blockDim.x + threadIdx.x; addr < n*n;  
       addr += blockDim.x * gridDim.x) {
    int i = (addr&mask1) + ((addr&mask3)>>shift3);
    int j = ((addr&mask2)>>shift2) + ((addr&mask4)>>shift4);

    Real sum = 0;
#pragma unroll 1024
    for (int k = 0; k < n; ++k) {
      sum += ma[k*n+i]*ma[k*n+j];
    }
    mc[i*n+j] = sum;
  }
}

void benchmark (const int logN, const int logNB) {


  const int n  = 1<<logN;
  

  
  thrust::device_vector<Real> ma(n*n), mb(n*n), mc(n*n);
  thrust::host_vector<Real> mh(n*n);
  for (int addr = 0; addr < n*n; ++addr) {
    mh[addr] = Real(1);
  }
  ma = mh; mb = mh;

  hipDeviceSynchronize();double time_begin = get_time<double>();
  calculate<<<1024, 448*2>>>
    (logN, logNB,
     thrust::raw_pointer_cast(&*ma.begin()),
     thrust::raw_pointer_cast(&*mb.begin()),
     thrust::raw_pointer_cast(&*mc.begin())
     );
  hipDeviceSynchronize();double time_end = get_time<double>();
  
  mh = mc;
  bool correct = true;
  for (int addr = 0; addr < n*n; ++addr) {
    if (abs(mh[addr]-n) > 0.1) {
      correct = false; break;
    }
  }

  
  double flop = double(n)*n*n*2;
  double time_cost = time_end - time_begin;
  double flops = flop / time_cost;
  long long int score = correct ? flops : 0;
  cout << score << "\t| "
       << correct << " " << logN << " " << logNB << " " << realTypename(Real(0)) << " : "
       << flops/1e9 << " Gflops=  " << flop << " / " << time_cost << endl;
}

int main () {
  // Set preference for above kernel to L1
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( calculate), hipFuncCachePreferL1 );
  
  for (int logN = 4; logN <= 12; ++logN) {
    for (int logNB = 1; logNB < logN; ++logNB) {
      benchmark(logN, logNB);
    }
  }
}


/*
int main () {
  for (int logN = 4; logN <= 13; ++logN) {
    for (int logNB = 1; logNB < logN; ++logNB) {
      const int n = 1<<logN;
      const int mask1  = (1<<logNB)-1;
      const int shift2 = logNB;
      const int mask2  = ((1<<logNB)-1)<<logNB;
      const int shift3 = logNB;
      const int mask3  = ((1<<(logN-logNB))-1)<<(2*logNB);
      const int shift4 = logN;
      const int mask4  = ((1<<(logN-logNB))-1)<<(logNB+logN);

      cerr << mask1 << " " << mask2 << " " << mask3 << " " << mask4 << endl;
      
      for (int addr = 0; addr < n*n; ++addr) {
	int i = (addr&mask1) + ((addr&mask3)>>shift3);
	int j = ((addr&mask2)>>shift2) + ((addr&mask4)>>shift4);

	cout << i << " " << j << endl;
      }
      return 0;
    }
  }
}

*/
