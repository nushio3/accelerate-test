#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
using namespace std;

#include "get_time.h"

typedef float Real;

__global__
void calculate (const int n, Real *ma, Real *mb, Real *mc) {
  for (int addr = blockIdx.x * blockDim.x + threadIdx.x; addr < n*n;  
       addr += blockDim.x * gridDim.x) {
    int i = addr%n;
    int j = addr/n;

    Real sum = 0;
#pragma unroll 32
    for (int k = 0; k < n; ++k) {
      sum += ma[i*n+k]*ma[j*n+k]
    }
    mc[i*n+j] = sum;
  }
}

void benchmark (const int n) {

  double time_begin = get_time<double>();
  
  thrust::device_vector<Real> ma(n*n), mb(n*n), mc(n*n);
  thrust::host_vector<Real> mh(n*n);
  for (int addr = 0; addr < n*n; ++addr) {
    mh[addr] = Real(1);
  }
  ma = mh; mb = mh;
  calculate<Iter><<<1024, 448>>>
    (n,
     thrust::raw_pointer_cast(&*ma.begin()),
     thrust::raw_pointer_cast(&*mb.begin()),
     thrust::raw_pointer_cast(&*mc.begin())
     );
  mh = mc;
  bool correct = true;
  for (int addr = 0; addr < n*n; ++addr) {
    if (abs(mh[addr]-n) > 0.1) {
      correct = false; break;
    }
  }

  double time_end = get_time<double>();
  
  double flop = double(n)*n*n*2;
  double time_cost = time_end - time_begin;
  double flops = flop / time_cost;
  cout << correct << " " << flops/1e9 << " Gflops =  "
       << flop << " / " << time_cost << endl;
}

int main () {
  for (int n = 1; n <= 1<<12; n*=2)
    benchmark(n);
}

