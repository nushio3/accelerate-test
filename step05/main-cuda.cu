#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <string>
#include <stdlib.h>
#include <vector>
using namespace std;

typedef float Real;

#include "fluid.h"
#include "get_time.h"

int zoom;
Real flowSpeed;

__global__ void initialize (Real flowSpeed, FluidPtr pFlu) {
  pFlu.initialize(flowSpeed);
}

__global__ void collision (FluidPtr pFlu, FluidPtr pFlu2) {
  pFlu.collision(pFlu2);
}

__global__ void proceed (FluidPtr pFlu, FluidPtr pFlu2) {
  pFlu2.proceed(pFlu);
}



int main (int argc, char **argv) {
  if (argc < 3) {
    cerr << "usage : " << argv[0] << " zoom flowSpeed" << endl;
    return -1;
  } else {
    istringstream iss(argv[1]);
    iss >> zoom;
    istringstream iss2(argv[2]);
    iss2 >> flowSpeed;
  }

  cerr << argv[0] << endl;
  
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(collision), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(proceed  ), hipFuncCachePreferL1);

  string dirn;
  {
    ostringstream oss; oss << "bin/" << zoom << "_" << flowSpeed;
    dirn = oss.str();
    system(("mkdir -p " + dirn).c_str());
  }

  FluidMemory<thrust::device_vector<Real> > flu(1024*zoom,768*zoom);
  FluidMemory<thrust::host_vector<Real> > flu_host(1024*zoom,768*zoom);
  FluidMemory<thrust::device_vector<Real> >  flu2=flu;

  FluidPtr pFlu = flu.ptr();
  FluidPtr pFlu2 = flu2.ptr();

  const int blockDim = 1024, gridDim = 448;

  initialize<<<blockDim,gridDim>>>(flowSpeed, pFlu);
  initialize<<<blockDim,gridDim>>>(flowSpeed, pFlu2);

  double time_integrated = 0;

  const int max_t = 50001; //100001;
  for (int t = 0; t < zoom*max_t; ++t) {
    if (t % (zoom*50000) == 0) {
      ostringstream ossFn;
      ossFn << dirn << "/" << (100000000+t) << ".bin";
      cerr << ossFn.str() << " : time spent so far " << time_integrated << endl;
      flu_host.copyFrom(flu);
      flu_host.write(ossFn.str(), zoom);
    }

    double time_begin = get_time<double>();
    hipDeviceSynchronize();
    collision<<<blockDim,gridDim>>>(pFlu, pFlu2);
    hipDeviceSynchronize();
    proceed<<<blockDim,gridDim>>>(pFlu, pFlu2);
    hipDeviceSynchronize();
    double time_end = get_time<double>();
    time_integrated += time_end - time_begin;
  }
  
  return 0;
}
