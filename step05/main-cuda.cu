#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <string>
#include <stdlib.h>
#include <vector>
using namespace std;

typedef float Real;

#include "fluid.h"

int zoom;
Real flowSpeed;

__global__ void initialize (Real flowSpeed, FluidPtr pFlu) {
  pFlu.initialize(flowSpeed);
}

__global__ void collision (FluidPtr pFlu, FluidPtr pFlu2) {
  pFlu.collision(pFlu2);
}

__global__ void proceed (FluidPtr pFlu, FluidPtr pFlu2) {
  pFlu2.proceed(pFlu);
}



int main (int argc, char **argv) {
  if (argc < 3) {
    cerr << "usage : " << argv[0] << " zoom flowSpeed" << endl;
    return -1;
  } else {
    istringstream iss(argv[1]);
    iss >> zoom;
    istringstream iss2(argv[2]);
    iss2 >> flowSpeed;
  }

  string dirn;
  {
    ostringstream oss; oss << "bin/" << zoom << "_" << flowSpeed;
    dirn = oss.str();
    system(("mkdir -p " + dirn).c_str());
  }

  FluidMemory<thrust::device_vector<Real> > flu(1024*zoom,768*zoom);
  FluidMemory<thrust::device_vector<Real> >  flu2=flu;

  FluidPtr pFlu = flu.ptr();
  FluidPtr pFlu2 = flu2.ptr();

  initialize<<<1024,448>>>(flowSpeed, pFlu);
  initialize<<<1024,448>>>(flowSpeed, pFlu2);
  
  for (int t = 0; t < zoom*100001; ++t) {
    if (t % (zoom*100) == 0) {
      ostringstream ossFn;
      ossFn << dirn << "/" << (100000000+t) << ".bin";
      cerr << ossFn.str() << endl;
      flu.write(ossFn.str(), zoom);
    }

    collision<<<1024,448>>>(pFlu, pFlu2);
    proceed<<<1024,448>>>(pFlu, pFlu2);
  }
  
  return 0;
}
